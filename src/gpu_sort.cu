// CUDA/Thrust GPU sort for large arrays of record_t by key.
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include "record.h"

extern "C" void gpu_sort_records(record_t* host_arr, size_t n) {
    // copy keys and values separately to device
    thrust::device_vector<long long> d_keys(n);
    thrust::device_vector<double>    d_vals(n);
    for (size_t i = 0; i < n; ++i) {
        d_keys[i] = (long long)host_arr[i].key;
        d_vals[i] = host_arr[i].val;
    }
    thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_vals.begin());
    // copy back
    for (size_t i = 0; i < n; ++i) {
        host_arr[i].key = (long long)d_keys[i];
        host_arr[i].val = (double)d_vals[i];
    }
}